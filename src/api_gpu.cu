#include "utils.h"
#include "utils_gpu.h"
#include "api_gpu.h"

inline int getStreamNum(size_t fftSize)
{
    size_t avail;
    size_t total;
    hipMemGetInfo(&avail,&total);
    size_t ret=(avail-16*fftSize)/(40*fftSize+10*1024*1024);
    return (int)ret;
}

void init_gpu(int fftSize,int pulseNumber,int sampleNumber,double parameters1,double parameters2,
	int windowType1,int windowType2,float *freqWeight)
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp,dev));
    printf("Using device %d: %s\n",dev,deviceProp.name);
    CHECK(hipSetDevice(dev));

    streamNum=getStreamNum(fftSize);
    if(streamNum==0)
    {
        puts("ERROR:The GPU memory is not enough.");
        exit(0);
    }
    if(setStreamNum!=0)
    {
        if(setStreamNum>streamNum)
        {
            puts("ERROR:The Manual Stream Number is Too Much.");
            exit(0);
        }
        streamNum=setStreamNum;
    }
    printf("GPU streamNum : %d\n",streamNum);
    stream=(hipStream_t *)fftw_malloc(streamNum*sizeof(hipStream_t));
    for(int i=0;i<streamNum;i++)
        hipStreamCreateWithFlags(&stream[i],hipStreamDefault/*hipStreamNonBlocking*/);

    CHECK(hipMalloc((void**)&pingBuf,sizeof(hipfftComplex)*fftSize*streamNum));
    CHECK(hipMemset(pingBuf,0,sizeof(hipfftComplex)*fftSize*streamNum));
    CHECK(hipMalloc((void**)&pongBuf,sizeof(hipfftComplex)*fftSize*streamNum));
    CHECK(hipMemset(pongBuf,0,sizeof(hipfftComplex)*fftSize*streamNum));
    CHECK(hipMalloc((void**)&cfar_gpu,sizeof(float)*pulseNumber*sampleNumber*streamNum));
    CHECK(hipMalloc((void**)&threshold_gpu,sizeof(float)*pulseNumber*sampleNumber*streamNum));

    CHECK(hipMalloc((void**)&d_block_sums,sizeof(float)*(fftSize/1024+1)*streamNum));
    // CHECK(hipMemset(d_block_sums,0,sizeof(float)*(fftSize/1024+1)*streamNum));
    CHECK(hipMalloc((void**)&d_in_block_sums,sizeof(float)*(fftSize/1024+1)*streamNum));
    // CHECK(hipMemset(d_in_block_sums,0,sizeof(float)*(fftSize/1024+1)*streamNum));
    CHECK(hipMalloc((void**)&d_dummy_blocks_sums,sizeof(float)*streamNum));
    // CHECK(hipMemset(d_dummy_blocks_sums,0,sizeof(float)*streamNum));


    plan1=(hipfftHandle *)fftw_malloc(sizeof(hipfftHandle)*streamNum);
    plan2=(hipfftHandle *)fftw_malloc(sizeof(hipfftHandle)*streamNum);
    handle=(hipblasHandle_t *)fftw_malloc(sizeof(hipblasHandle_t)*streamNum);
    for(int i=0;i<streamNum;i++)
    {
        hipfftPlan1d(&plan1[i],fftSize,HIPFFT_C2C,1);
        hipfftPlan1d(&plan2[i],pulseNumber-1,HIPFFT_C2C,sampleNumber);
        hipblasCreate(&handle[i]);
        hipfftSetStream(plan1[i],stream[i]);
        hipfftSetStream(plan2[i],stream[i]);
        hipblasSetStream(handle[i],stream[i]);
    }

    CHECK(hipMalloc((void**)&window1,sizeof(double)*fftSize));
    CHECK(hipMalloc((void**)&window2,sizeof(double)*pulseNumber));
    double *window_func=(double *)fftw_malloc(fftSize*sizeof(double));
    create_window(window_func,fftSize,windowType1,parameters1);
    CHECK(hipMemcpy(window1,window_func,sizeof(double)*fftSize,hipMemcpyHostToDevice));
    fftw_free(window_func);
    window_func=(double *)fftw_malloc(pulseNumber*sizeof(double));
    create_window(window_func,pulseNumber,windowType2,parameters2);
    CHECK(hipMemcpy(window2,window_func,sizeof(double)*pulseNumber,hipMemcpyHostToDevice));
    fftw_free(window_func);
    CHECK(hipMalloc((void**)&freqWeight_gpu,sizeof(float)*pulseNumber*sampleNumber));
    CHECK(hipMemcpy(freqWeight_gpu,freqWeight,sizeof(float)*pulseNumber*sampleNumber,hipMemcpyHostToDevice));
}

void release_gpu()
{
    CHECK(hipFree(pingBuf));
    CHECK(hipFree(pongBuf));
    CHECK(hipFree(cfar_gpu));
    CHECK(hipFree(threshold_gpu));

    CHECK(hipFree(d_block_sums));
    CHECK(hipFree(d_in_block_sums));
    CHECK(hipFree(d_dummy_blocks_sums));

    for(int i=0;i<streamNum;i++)
    {
        hipStreamDestroy(stream[i]);
        hipfftDestroy(plan1[i]);
        hipfftDestroy(plan2[i]);
        hipblasDestroy(handle[i]);
    }

    CHECK(hipFree(window1));
    CHECK(hipFree(window2));
    CHECK(hipFree(freqWeight_gpu));

    fftw_free(stream);
    fftw_free(plan1);
    fftw_free(plan2);
    fftw_free(handle);
}

void malloc_IO_memory_gpu(fftwf_complex **signalAll,fftwf_complex **coeff,float **cfar,float **threshold)
{
    hipHostMalloc((fftwf_complex**)signalAll,totalNumber*sizeof(fftwf_complex));
    hipHostMalloc((fftwf_complex**)coeff,blindNumber*sizeof(fftwf_complex));
    hipHostMalloc((float**)cfar,(totalNumber)*sizeof(float));
    hipHostMalloc((float**)threshold,(totalNumber)*sizeof(float));
}

void free_IO_memory_gpu(fftwf_complex *signalAll,fftwf_complex *coeff,float *cfar,float *threshold)
{
    hipHostFree(coeff);
    hipHostFree(signalAll);
    hipHostFree(cfar);
    hipHostFree(threshold);
}

void sync_time_gpu()
{
    hipDeviceSynchronize();
    _end=cpuSecond();
    double timeCost=_end-start;
	totalTime+=timeCost;
}